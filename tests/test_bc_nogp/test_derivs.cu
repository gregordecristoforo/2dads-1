#include "hip/hip_runtime.h"
/*
 * Test derivatives
 *
 * Input:
 *      f(x, y) = sin(2.0 * pi * x)
 *      f_x = 2 * pi * cos(2 pi x)
 *      -> Initializes arr1
 *
 *      g(x, y) = cos(2.0 * pi * y)
 *      g_y = -2 pi sin(y)
 *      -> Initializes arr2
 *
 *
 *
 */

#include <iostream>
#include <fstream>
#include <sstream>
#include "slab_bc.h"

using namespace std;

int main(void)
{
    size_t Nx{16};
    size_t My{16};
    cout << "Enter Nx: ";
    cin >> Nx;
    cout << "Enter My: ";
    cin >> My;

    stringstream fname;
    ofstream of;

    cuda::bvals_t<double> my_bvals{cuda::bc_t::bc_dirichlet, cuda::bc_t::bc_dirichlet, cuda::bc_t::bc_periodic, cuda::bc_t::bc_periodic, 0.0, 0.0, 0.0, 0.0};
    cuda::slab_layout_t my_geom(0.0, 1.0 / double(Nx), 0.0, 1.0 / double(My), Nx, 0, My, 2);

    {
        slab_bc my_slab(my_geom, my_bvals);
        my_slab.initialize_derivatives(test_ns::field_t::arr1, test_ns::field_t::arr2);

        // Initialize analytic solution
        cuda_array_bc_nogp<my_allocator_device<cuda::real_t>> sol_an(my_geom, my_bvals, 1);
        sol_an.evaluate([=] __device__(size_t n, size_t m, cuda::slab_layout_t geom) -> cuda::real_t
            {
                cuda::real_t x{geom.get_xleft() + (cuda::real_t(n) + 0.5) * geom.get_deltax()};
                cuda::real_t y{geom.get_ylo() + (cuda::real_t(m) + 0.5) * geom.get_deltay()};
                return(cuda::TWOPI * cos(cuda::TWOPI * x));
            }, 0);

        // Write analytic solution to file
        fname.str(string(""));
        fname << "test_derivs_ddx1_solan_" << Nx << "_out.dat";
        of.open(fname.str());
        of << sol_an;
        of.close();

        //// compute first x-derivative
        my_slab.d_dx(test_ns::field_t::arr1, test_ns::field_t::arr3, 1, 0, 0);

        fname.str(string(""));
        fname << "test_derivs_ddxx1_solnum_" << Nx << "_out.dat";
        my_slab.print_field(test_ns::field_t::arr3, fname.str());

        cuda_array_bc_nogp<my_allocator_device<cuda::real_t>> sol_num(my_slab.get_array_ptr(test_ns::field_t::arr3));
        sol_num -= sol_an;
        cout << "sol_num - sol_an: Nx = " << Nx << ", My = " << My << ", L2 = " << sol_num.L2(0) << endl;
    }
}
