#include "hip/hip_runtime.h"
/*
 * Test Arakawa bracket operators
 * Compute {f,g} = -f_y g_x + g_y f_x
 * 
 * Input:
 *     f(x, y) = -sin(2 pi x)^2 sin(2 pi y)^2
 *     f_x = -4 pi (cos 2 pi x)sin(2 pi x) sin(2 pi y)^2
 *     f_y = -4 pi(cos 2 pi y) sin(2 pi y) sin(2 pi x)^2
 *     -> initializes arr1
 * 
 *     g(x, y) = sin(pi x) sin(pi y)
 *     g_x = pi cos(pi x) sin(pi y)
 *     g_y = pi sin(pi x) cos(pi y)
 *     -> initializes arr2
 *
 * Output
 *     {f,g} = 16 pi^2 cos(pi x) cos(pi y) [-(cos(2 pi x) + cos(2 pi y))sin (pi x)^2 sin(pi y)^2
 *     -> stored in arr3
 *
 *
 */

#include <iostream>
#include <sstream>
#include "slab_bc.h"

using namespace std;


int main(void){
    constexpr twodads::real_t x_l{-1.0};
    constexpr twodads::real_t Lx{2.0};
    constexpr twodads::real_t y_l{-1.0};
    constexpr twodads::real_t Ly{2.0};

    size_t Nx{128};
    size_t My{128};
    cout << "Enter Nx: ";
    cin >> Nx;
    cout << "Enter My: ";
    cin >> My;

    stringstream fname;
    ofstream of;

    twodads::slab_layout_t my_geom(x_l, Lx / twodads::real_t(Nx), y_l, Ly / twodads::real_t(My), Nx, 0, My, 2, twodads::grid_t::cell_centered);
    twodads::bvals_t<double> my_bvals{twodads::bc_t::bc_dirichlet, twodads::bc_t::bc_dirichlet, twodads::bc_t::bc_periodic, twodads::bc_t::bc_periodic,
        0.0, 0.0, 0.0, 0.0};
    twodads::stiff_params_t stiff_params(0.1, Lx, Ly, 0.1, 0.0, Nx, My / 2 + 1, 4);

    {
        std::cout << "helleo?" << std::endl;
        slab_bc my_slab(my_geom, my_bvals, stiff_params);
        std::cout << "slab created" << std::endl;
        cuda_array_bc_nogp<twodads::real_t, allocator_device> sol_an(my_geom, my_bvals, 1);
        sol_an.apply([] __device__ (twodads::real_t dummy, size_t n, size_t m, twodads::slab_layout_t geom) -> twodads::real_t
                {
                    twodads::real_t x{geom.get_x(n)};
                    twodads::real_t y{geom.get_y(m)};
                    return(16.0 * twodads::PI * twodads::PI * cos(twodads::PI * x) * cos(twodads::PI * y) * (cos(twodads::TWOPI * x) - cos(twodads::TWOPI * y)) * sin(twodads::PI * x) * sin(twodads::PI * x) * sin(twodads::PI * y) * sin(twodads::PI * y));
                }, 
                0);

        fname.str(string(""));
        fname << "test_arakawa_solan_" << Nx << "_out.dat";
        utility :: print(sol_an, 0, fname.str());

        my_slab.initialize_arakawa(test_ns::field_t::arr1, test_ns::field_t::arr2, 0);
        // Print input to inv_laplace routine into array arr1_nx.dat
        fname.str(string(""));
        fname << "test_arakawa_f_" << Nx << "_in.dat";
        utility :: print((*my_slab.get_array_ptr(test_ns::field_t::arr1)), 0, fname.str());

        fname.str(string(""));
        fname << "test_arakawa_g_" << Nx << "_in.dat";
        utility :: print((*my_slab.get_array_ptr(test_ns::field_t::arr2)), 0, fname.str());

        std::cout << "computing poisson bracket" << std::endl;
        my_slab.arakawa(test_ns::field_t::arr1, test_ns::field_t::arr2, test_ns::field_t::arr3, size_t(0), size_t(0));

        fname.str(string(""));
        fname << "test_arakawa_solnum_" << Nx << "_out.dat";
        utility :: print((*my_slab.get_array_ptr(test_ns::field_t::arr3)), 0, fname.str());
       
        cuda_array_bc_nogp<twodads::real_t, allocator_device> sol_num(my_slab.get_array_ptr(test_ns::field_t::arr3));
        sol_num -= sol_an;

        cout << "sol_num - sol_an: Nx = " << Nx << ", My = " << My << ", L2 = " << utility :: L2(sol_num, 0) << endl;

        fname.str(string(""));
        fname << "test_arakawa_diff_" << Nx << "_out.dat";
        utility :: print(sol_num, 0, fname.str());
    }
    hipDeviceReset();
}

// End of file test_arakawa.cu
