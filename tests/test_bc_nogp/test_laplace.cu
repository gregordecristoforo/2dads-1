#include "hip/hip_runtime.h"
/*
 * Invert the laplace equation with boundary conditions in x using the cusparse tridiagonal solver with the new datatype
 *
 * Invert
 * g(x,y) = exp(-(x^2 + y^2) / 2)
 * \nabla^2 g(x,y) = f(x,y) 
 * where
 * f(x,y) = exp(-(x^2 + y^2) / 2) (-2 + x^2 + y^2)
 *
 * Goal: Given f(x,y) find g(x,y)
 */


#include <iostream>
#include <sstream>
#include "slab_bc.h"

using namespace std;

int main(void){
    size_t Nx{128};
    size_t My{128};
    constexpr twodads::real_t x_l{-10.0};
    constexpr twodads::real_t Lx{20.0};
    constexpr twodads::real_t y_l{-10.0};
    constexpr twodads::real_t Ly{20.0};
    constexpr size_t tlevs{1};         // time levels for all arrays
    constexpr size_t tsrc{0};          // the time level we operate on

    cout << "Enter Nx: ";
    cin >> Nx;
    cout << "Enter My: ";
    cin >> My;

    stringstream fname;

    twodads::slab_layout_t my_geom(x_l, Lx / double(Nx), y_l, Ly / double(My), Nx, 0, My, 2, twodads::grid_t::cell_centered);
    twodads::bvals_t<double> my_bvals{twodads::bc_t::bc_dirichlet, twodads::bc_t::bc_dirichlet, twodads::bc_t::bc_periodic, twodads::bc_t::bc_periodic,
                                   0.0, 0.0, 0.0, 0.0};
    twodads::stiff_params_t params(0.001, 20.0, 20.0, 0.001, 0.0, my_geom.get_nx(), (my_geom.get_my() + my_geom.get_pad_y()) / 2, tlevs);
    {
        slab_bc my_slab(my_geom, my_bvals, params);
        my_slab.initialize_invlaplace(twodads::field_t::f_omega, tsrc);

        fname << "test_laplace_input_" << Nx << "_device.dat";
        utility :: print((*my_slab.get_array_ptr(twodads::field_t::f_omega)), tsrc, fname.str());

        cuda_array_bc_nogp<twodads::real_t, allocator_device> sol_an(my_geom, my_bvals, tlevs);
        sol_an.apply([] __device__ (twodads::real_t dummy, size_t n, size_t m, twodads::slab_layout_t geom) -> twodads::real_t
                {
                    const twodads::real_t x{geom.get_x(n)};
                    const twodads::real_t y{geom.get_y(m)};
                    return(exp(-0.5 * (x * x + y * y)));
                },
            tsrc);

        fname.str(string(""));
        fname << "test_laplace_solan_" << Nx << "_device.dat";
        utility :: print(sol_an, tsrc, fname.str());

        my_slab.invert_laplace(twodads::field_t::f_omega, twodads::field_t::f_strmf, tsrc, tsrc);

        // Write numerical solution to file
        fname.str(string(""));
        fname << "test_laplace_solnum_" << Nx << "_device.dat";
        utility :: print((*my_slab.get_array_ptr(twodads::field_t::f_strmf)), tsrc, fname.str());

        // Get the analytic solution
        sol_an -= my_slab.get_array_ptr(twodads::field_t::f_strmf);
        cout << "Nx = " << Nx << ", My = " << My << ", L2 = " << utility :: L2(sol_an, tsrc) << endl;
    } // Let managed memory go out of scope before calling hipDeviceReset()
    // However cublas_handle_t survivs this scoping and we get a segfault from its destructor
    /*
    warning: Cuda API error detected: hipFree returned (0x11)

warning: Cuda API error detected: hipEventDestroy returned (0x1e)

warning: Cuda API error detected: hipEventDestroy returned (0x1e)


Program received signal SIGSEGV, Segmentation fault.
0x00007fffeb3e903a in cuMemGetAttribute_v2 () from /usr/lib/x86_64-linux-gnu/libcuda.so.1
(cuda-gdb) bt
#0  0x00007fffeb3e903a in cuMemGetAttribute_v2 () from /usr/lib/x86_64-linux-gnu/libcuda.so.1
#1  0x00007fffeb3598a6 in cuVDPAUCtxCreate () from /usr/lib/x86_64-linux-gnu/libcuda.so.1
#2  0x00007fffeb33293a in hipEventDestroy () from /usr/lib/x86_64-linux-gnu/libcuda.so.1
#3  0x00007fffefad7724 in cublasSgemmEx () from /usr/local/cuda/lib64/libcublas.so.7.5
#4  0x00007fffefb0b984 in cublasSgemmEx () from /usr/local/cuda/lib64/libcublas.so.7.5
#5  0x00007fffef8e4797 in ?? () from /usr/local/cuda/lib64/libcublas.so.7.5
#6  0x00007fffef91932d in hipblasDestroy () from /usr/local/cuda/lib64/libcublas.so.7.5
#7  0x0000000000414e38 in solvers::cublas_handle_t::~cublas_handle_t (this=0x728100 <solvers::cublas_handle_t::get_handle()::h>, __in_chrg=<optimized out>) at /home/rku000/source/2dads/include/solvers.h:49
#8  0x00007fffec1d8b29 in secure_getenv () from /lib/x86_64-linux-gnu/libc.so.6
#9  0x00007fffec1d8b75 in exit () from /lib/x86_64-linux-gnu/libc.so.6
#10 0x00007fffec1c2b4c in __libc_start_main () from /lib/x86_64-linux-gnu/libc.so.6

*/
    //hipDeviceReset();
}