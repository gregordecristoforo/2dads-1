#include "hip/hip_runtime.h"
/*
 * test_perf_derivs.cu
 *
 *  Created on: Feb 12, 2015
 *      Author: rku000
 */


/*
 * Test different ways of computing derivation of a field
 *
 *
 * ---------> x direction (Nx/2+1 modes)
 * |
 * |
 * |
 * V
 * y-direction My modes (My / 2 positive, My / 2 - 1 negative
 *
 */

#include <iostream>
#include <fstream>
#include <cmath>
#include "cucmplx.h"
#include "cuda_array4.h"
#include "derivatives.h"

typedef CuCmplx<double> cmplx_t;

#define ELEM_PER_THREAD_T 1


using namespace std;



int main(void)
{
	constexpr int Nx{64};
	constexpr int My{64};
	constexpr double Lx{10.0};
	constexpr double Ly{10.0};
	constexpr double dx{Lx / Nx};
	constexpr double dy{Ly / My};

    cout << "Grid: Lx = " << Lx << ", Nx = " << Nx << ", dx = " << dx << endl;

    cuda::slab_layout_t sl{-0.5 * Lx, dx, -0.5 * Ly, dy, My, Nx};

    // Create derivs object
    derivs<double> der(sl);


	cuda_array<double> r_arr(1, My, Nx);
    cuda_array<double> r_arr_x(1, My, Nx);
	cuda_array<double> r_arr_y(1, My, Nx);

    cuda_array<double> r_arr_x2(1, My, Nx);
	cuda_array<double> r_arr_y2(1, My, Nx);

	double x{0.0};
    double y{0.0};
	for(int m = 0; m < My; m++)
	{
		y = - 0.5 * Ly + m * dy;
		for(int n = 0; n < Nx; n++)
		{
			x = -0.5 * Lx + n * dx;
			r_arr(0, m, n) = exp(-0.5 * x * x - 0.5 * y * y);
		}
	}
	r_arr.copy_host_to_device();

    der.d_dx1_dy1(r_arr, r_arr_x, r_arr_y);

    der.d_dx2_dy2(r_arr, r_arr_x2, r_arr_y2);

//
//	// Initialize cufft
//	hipfftResult err;
//    hipfftHandle plan_r2c;
//    hipfftHandle plan_c2r;
//    err = hipfftPlan2d(&plan_r2c, Nx, My, HIPFFT_D2Z);
//    err = hipfftPlan2d(&plan_c2r, Nx, My, HIPFFT_Z2D);
//
//	err = hipfftExecD2Z(plan_r2c, r_arr.get_array_d(), (hipfftDoubleComplex*) c_arr.get_array_d(0));
//	if(err != HIPFFT_SUCCESS)
//		throw;
//
//	// Method 1... generate k-map and run derivs in one kernel call
//	gen_k_map<My, Nx21><<<dim3(1, My), dim3(Nx21, 1)>>>(kmap.get_array_d(), two_pi_Lx, two_pi_Ly);
//	gpuErrchk(hipPeekAtLastError());
//
////	void d_dx_dy_map(cmplx_t*  in, cmplx_t*  out_x, cmplx_t*  out_y, cmplx_t*  kmap)
//
//	//d_dx_dy_map<My, Nx21, elem_per_thread><<<gridsize, blocksize>>>(c_arr.get_array_d(), c_arr_x.get_array_d(), c_arr_y.get_array_d(), kmap.get_array_d());
//	d_dx_dy_map_sh<My, Nx21, elem_per_thread><<<gridsize, blocksize, shmem_size>>>(c_arr.get_array_d(), c_arr_x.get_array_d(), c_arr_y.get_array_d(), kmap.get_array_d());
//
//	gpuErrchk(hipPeekAtLastError());
//
//
//	// Transform to real space
//	err = hipfftExecZ2D(plan_c2r, (hipfftDoubleComplex*) c_arr.get_array_d(), r_arr.get_array_d());
//	err = hipfftExecZ2D(plan_c2r, (hipfftDoubleComplex*) c_arr_x.get_array_d(), r_arr_x.get_array_d());
//	err = hipfftExecZ2D(plan_c2r, (hipfftDoubleComplex*) c_arr_y.get_array_d(), r_arr_y.get_array_d());
//
//	r_arr.normalize();
//	r_arr_x.normalize();
//	r_arr_y.normalize();
//
//
	// output
    ofstream of;
	of.open("r_arr.dat");
	of << r_arr;
	of.close();

	of.open("r_arr_x.dat");
	of << r_arr_x;
	of.close();

	of.open("r_arr_y.dat");
	of << r_arr_y;
	of.close();

	of.open("r_arr_x2.dat");
	of << r_arr_x2;
	of.close();

	of.open("r_arr_y2.dat");
	of << r_arr_y2;
	of.close();

/*
	for(int t = 0; t < 1000; t++)
	{
	    d_dx_dy_map_sh<My, Nx21, elem_per_thread><<<gridsize, blocksize, shmem_size>>>(c_arr.get_array_d(), c_arr_x.get_array_d(), c_arr_y.get_array_d(), kmap.get_array_d());
		if (t % 50 == 0)
			cout << t << endl;
	}
*/
}



