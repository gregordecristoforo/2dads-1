#include "hip/hip_runtime.h"
/*
 * Only functions that require their own kernel
 *
 */

#include "include/slab_cuda.h"
#include <algorithm> //std::min, std::max


/*****************************************************************************
 *
 * Kernel implementation
 *
 ****************************************************************************/

// d/dx: Frequencies 0 .. N/2 - 1
__global__
void d_d_dx_lo(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double Lx)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint index = row * My + col;

    double two_pi_L = cuda::TWOPI / Lx;
    // Return if we don't have an item to work on
    if((col >= My) || (row >= Nx))
        return;
    out[index] = hipCmul(in[index], make_hipDoubleComplex(0.0, two_pi_L * double(row)));
}


// Frequencies: Nx/2
__global__
void d_d_dx_mid(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double Lx)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    //const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint row = Nx / 2;
    const uint index = row * My + col;

    //const double two_pi_L = 2.0 * cuda::PI / Lx;
    // Return if we don't have an item to work on
    if((col >= My) || (row >= Nx))
        return;
    out[index] = make_hipDoubleComplex(0.0, 0.0);
}


// Frequencies: Nx/2 + 1 ... Nx - 1
__global__
void d_d_dx_up(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double Lx)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x + Nx / 2 + 1;
    const uint index = row * My + col;


    double two_pi_L = cuda::TWOPI / Lx;
    // Return if we don't have an item to work on
    if((col >= My) || (row >= Nx))
        return;
    out[index] = hipCmul(in[index], make_hipDoubleComplex(0.0, two_pi_L * (double(row) - double(Nx))));
}


// Frequencies 0..My / 2
__global__
void d_d_dy_lo(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double Ly)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint index = row * My + col;

    if ((col >= My) || (row >= Nx))
        return;
    double two_pi_L = cuda::TWOPI / Ly;

    out[index] = hipCmul(in[index], make_hipDoubleComplex(0.0, two_pi_L * double(col)));
}


__global__
void d_d_dy_up(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double Ly)
{
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= Nx)
        return;

    index = (index + 1) * My - 1;
    out[index] = hipCmul(in[index], make_hipDoubleComplex(0.0, 0.0));
}

//
//
// invert two dimensional laplace equation.
// In spectral space, 
//                              / 4 pi^2 ((kx/Lx)^2 + (ky/Ly)^2 )  for kx, ky  <= N/2
// phi(kx, ky) = omega(kx, ky)  
//                              / 4 pi^2 (((kx-Nx)/Lx)^2 + (ky/Ly)^2) for kx > N/2 and ky <= N/2
//
// and phi(0,0) = 0 (to avoid division by zero)
// Divide into 4 sectors:
//
//            My/2    1 (last element)
//         ^<------>|------|
//  Nx/2+1 |        |      |
//         |   I    | III  |
//         |        |      |
//         v        |      |
//         =================
//         ^        |      |
//         |        |      |
//  Nx/2-1 |  II    |  IV  |   
//         |        |      |
//         v<------>|------|
//           My/2      1
//
// 
// sector I    : kx <= Nx/2, ky <= My/2  BS = (1, cuda_blockdim_my), GS = (Nx/2+1, My / (2 * cuda_blockdim_my)
// sector II   : kx > Nx/2, ky <= My/2   BS = (1, cuda_blockdim_my), GS = (Nx/2-1, My / (2 * cuda_blockdim_my)
// sector III  : kx <= Nx/2, ky = My/2   BS = cuda_blockdim_nx, GS = ^(Nx / 2 + 1) / cuda_blockdim_nx^ (round up, thread returns if out of bounds)
// sector IV   : kx > Nx/2, ky = My/2    BS = cuda_blockdim_nx, GS = ^(Nx / 2 - 1) / cuda_blockdim_nx^ (round up, thread returns if out of bounds)
//
// Pro: wavenumbers can be computed from index without if-else blocks
// Con: Diverging memory access

__global__
//void d_inv_laplace_sec1(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double inv_Lx2, double inv_Ly2, double four_pi_s)
void d_inv_laplace_sec1(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double inv_Lx2, double inv_Ly2)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
        return;

    double factor = -cuda::FOURPIS * (double(row * row) * inv_Lx2 + double(col * col) * inv_Ly2);
    out[idx] = hipCdiv(in[idx], make_hipDoubleComplex(factor, 0.0));
    //out[idx] = make_hipDoubleComplex(double(row), double(col));
}


__global__
void d_inv_laplace_sec2(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double inv_Lx2, double inv_Ly2) 
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x + Nx / 2 + 1;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
        return;
    double factor = -cuda::FOURPIS * (
            ((double(row) - double(Nx)) * (double(row) - double(Nx))) * inv_Lx2 +
            (double(col * col) * inv_Ly2));
    out[idx] = hipCdiv(in[idx], make_hipDoubleComplex(factor, 0.0));
    //out[idx] = make_hipDoubleComplex(double(row) - double(Nx), double(col));
}


__global__
void d_inv_laplace_sec3(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double inv_Lx2, double inv_Ly2)
{
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = (row + 1) * My - 1;

    if (row > Nx / 2 + 1)
        return;

    double factor = -cuda::FOURPIS * (
            (double(row * row) * inv_Lx2 + double(My * My) * inv_Ly2));
    out[idx] = hipCdiv(in[idx], make_hipDoubleComplex(factor, 0.0));
    //out[idx] = make_hipDoubleComplex(double(row), double(My));
}


__global__
void d_inv_laplace_sec4(cuda::cmplx_t* in, cuda::cmplx_t* out, uint Nx, uint My, double inv_Lx2, double inv_Ly2) 
{
    const uint row = blockIdx.x * blockDim.x + threadIdx.x + Nx / 2 + 1;
    const uint idx = (row + 1) * My - 1;

    if (row >= Nx)
        return;

    double factor = -cuda::FOURPIS * (
            ((double(row) - double(Nx)) * (double(row) - double(Nx)) * inv_Lx2 + double(My * My) * inv_Ly2));
    out[idx] = hipCdiv(in[idx], make_hipDoubleComplex(factor, 0.0));
    //out[idx] = make_hipDoubleComplex(double(row) - double(Nx), double(My));
}


__global__
void d_inv_laplace_zero(cuda::cmplx_t* out)
{
    out[0] = make_hipDoubleComplex(0.0, 0.0);
}


/*
 * Stiffly stable time integration
 * temp = sum(k=1..level) alpha[T-2][level-k] * u^{T-k} + delta_t * beta[T-2][level -k - 1] * u_RHS^{T-k-1}
 * u^{n+1}_{i} = temp / (alpha[T-2][0] + delta_t * diff * (kx^2 + ky^2))
 *
 * Use same sector splitting as for inv_laplace
 */


__global__
void d_integrate_stiff_sec1(cuda::cmplx_t** A, cuda::cmplx_t** A_rhs, cuda::real_t* alpha, cuda::real_t* beta, cuda::stiff_params_t p, uint tlev) 
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * p.My + col;
    if ((col >= p.My) || (row >= p.Nx))
        return;

    unsigned int off_a = (tlev - 2) * p.level + tlev;
    unsigned int off_b = (tlev - 2) * (p.level - 1) + tlev- 1;
    cuda::real_t kx = cuda::real_t(row) * cuda::TWOPI / p.length_x;
    cuda::real_t ky = cuda::real_t(col) * cuda::TWOPI / p.length_y;
    cuda::cmplx_t sum1_alpha = make_hipDoubleComplex(0.0, 0.0);
    cuda::cmplx_t sum1_beta = make_hipDoubleComplex(0.0, 0.0);
    cuda::real_t temp_div = 1. / (alpha[(tlev - 2) * p.level] + p.delta_t * p.diff * (kx * kx + ky * ky));

    // Add contribution from explicit / implicit parts
    for(uint k = 1; k < tlev; k++)
    {
        //sum1_alpha = hipCadd(sum1_alpha, hipCmul(A[p.level - k][idx], alpha[off_a - k]));
        //sum1_beta = hipCadd(sum1_beta, hipCmul(A_rhs[p.level - 1 - k][idx], beta[off_b - k]));
        sum1_alpha.x += A[p.level - k][idx].x * alpha[off_a - k];
        sum1_alpha.y += A[p.level - k][idx].y * alpha[off_a - k];
        sum1_beta.x += A_rhs[p.level - 1 - k][idx].x * beta[off_b - k];
        sum1_beta.y += A_rhs[p.level - 1 - k][idx].y * beta[off_b - k];
    }
    //sum1_beta = hipCmul(sum1_beta, make_hipDoubleComplex(p.delta_t, 0.0));
    //A[p.level - tlev][idx] = hipCmul(hipCadd(sum1_alpha, sum1_beta), temp_div);
    A[p.level - tlev][idx].x = (sum1_alpha.x + p.delta_t * sum1_beta.x) * temp_div; 
    A[p.level - tlev][idx].y = (sum1_alpha.y + p.delta_t * sum1_beta.y) * temp_div; 
}


__global__
void d_integrate_stiff_sec2(cuda::cmplx_t** A, cuda::cmplx_t** A_rhs, cuda::real_t* alpha, cuda::real_t* beta, cuda::stiff_params_t p, uint tlev)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x + p.Nx / 2 + 1;
    const uint idx = row * p.My + col;
    if ((col >= p.My) || (row >= p.Nx))
        return;

    uint off_a = (tlev - 2) * p.level + tlev;
    uint off_b = (tlev - 2) * (p.level - 1) + tlev - 1;
    cuda::real_t kx = (cuda::real_t(row) - cuda::real_t(p.Nx)) * cuda::TWOPI / p.length_x;
    cuda::real_t ky = cuda::real_t(col) * cuda::TWOPI / p.length_y;
    cuda::cmplx_t sum1_alpha = make_hipDoubleComplex(0.0, 0.0);
    cuda::cmplx_t sum1_beta = make_hipDoubleComplex(0.0, 0.0);
    //cuda::cmplx_t temp_div = make_hipDoubleComplex(1. / (alpha[(tlev - 2) * p.level].x + p.delta_t * p.diff * (kx * kx + ky * ky)), 0.0);
    cuda::real_t temp_div = 1. / (alpha[(tlev - 2) * p.level] + p.delta_t * p.diff * (kx * kx + ky * ky));

    for(uint k = 1; k < tlev; k++)
    {
        //sum1_alpha = hipCadd(sum1_alpha, hipCmul(A[p.level - k][idx], alpha[off_a - k]));
        //sum1_beta = hipCadd(sum1_beta, hipCmul(A_rhs[p.level - 1 - k][idx], beta[off_b - k]));
        sum1_alpha.x += A[p.level - k][idx].x * alpha[off_a - k];
        sum1_alpha.y += A[p.level - k][idx].y * alpha[off_a - k];
        sum1_beta.x += A_rhs[p.level - 1 - k][idx].x * beta[off_b - k];
        sum1_beta.y += A_rhs[p.level - 1 - k][idx].y * beta[off_b - k];
    }
    //sum1_beta = hipCmul(sum1_beta, make_hipDoubleComplex(p.delta_t, 0.0));
    //A[p.level - tlev][idx] = hipCmul(hipCadd(sum1_alpha, sum1_beta), temp_div);
    A[p.level - tlev][idx].x = (sum1_alpha.x + p.delta_t * sum1_beta.x) * temp_div;
    A[p.level - tlev][idx].y = (sum1_alpha.y + p.delta_t * sum1_beta.y) * temp_div;
}


__global__
void d_integrate_stiff_sec3(cuda::cmplx_t** A, cuda::cmplx_t** A_rhs, cuda::real_t* alpha, cuda::real_t* beta, cuda::stiff_params_t p, uint tlev)
{
    const uint col = p.My;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = (row + 1) * p.My - 1; 
    if (row >= p.Nx)
        return;

    uint off_a = (tlev - 2) * p.level + tlev;
    uint off_b = (tlev - 2) * (p.level - 1) + tlev - 1;
    cuda::real_t kx = cuda::real_t(row) * cuda::TWOPI / p.length_x;
    cuda::real_t ky = cuda::real_t(col) * cuda::TWOPI/ p.length_y;
    cuda::cmplx_t sum1_alpha = make_hipDoubleComplex(0.0, 0.0);
    cuda::cmplx_t sum1_beta = make_hipDoubleComplex(0.0, 0.0);
    cuda::real_t temp_div = 1. / (alpha[(tlev - 2) * p.level] + p.delta_t * p.diff * (kx * kx + ky * ky));

    for(uint k = 1; k < tlev; k++)
    {
        //sum1_alpha = hipCadd(sum1_alpha, hipCmul(A[p.level - k][idx], alpha[off_a - k]));
        //sum1_beta = hipCadd(sum1_beta, hipCmul(A_rhs[p.level - 1 - k][idx], beta[off_b - k]));
        sum1_alpha.x += A[p.level - k][idx].x * alpha[off_a - k];
        sum1_alpha.y += A[p.level - k][idx].y * alpha[off_a - k];
        sum1_beta.x += A_rhs[p.level - 1 - k][idx].x * beta[off_b - k];
        sum1_beta.y += + A_rhs[p.level - 1 - k][idx].y * beta[off_b - k];
    }
    //sum1_beta = hipCmul(sum1_beta, make_hipDoubleComplex(p.delta_t, 0.0));
    //A[p.level - tlev][idx] = hipCmul(hipCadd(sum1_alpha, sum1_beta), temp_div);
    A[p.level - tlev][idx].x = (sum1_alpha.x + p.delta_t * sum1_beta.x) * temp_div;
    A[p.level - tlev][idx].y = (sum1_alpha.y + p.delta_t * sum1_beta.y) * temp_div;
}


__global__
void d_integrate_stiff_sec4(cuda::cmplx_t** A, cuda::cmplx_t** A_rhs, cuda::real_t* alpha, cuda::real_t* beta, cuda::stiff_params_t p, uint tlev)
{
    const uint col = p.My;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x + p.Nx / 2 + 1;
    const uint idx = (row + 1) * p.My - 1; 
    if (row >= p.Nx)
        return;

    uint off_a = (tlev - 2) * p.level + tlev;
    uint off_b = (tlev - 2) * (p.level - 1) + tlev - 1;
    cuda::real_t kx = (cuda::real_t(row) - cuda::real_t(p.Nx)) * cuda::TWOPI / p.length_x;
    cuda::real_t ky = cuda::real_t(col) * cuda::TWOPI / p.length_y;
    cuda::cmplx_t sum1_alpha = make_hipDoubleComplex(0.0, 0.0);
    cuda::cmplx_t sum1_beta = make_hipDoubleComplex(0.0, 0.0);
    cuda::real_t temp_div = 1. / (alpha[(tlev - 2) * p.level] + p.delta_t * p.diff * (kx * kx + ky * ky));

    for(uint k = 1; k < tlev; k++)
    {
        //sum1_alpha = hipCadd(sum1_alpha, hipCmul(A[p.level - k][idx], alpha[off_a - k]));
        //sum1_beta = hipCadd(sum1_beta, hipCmul(A_rhs[p.level - 1 - k][idx], beta[off_b - k]));
        sum1_alpha.x += A[p.level - k][idx].x * alpha[off_a - k];
        sum1_alpha.y += A[p.level - k][idx].y * alpha[off_a - k];
        sum1_beta.x += A_rhs[p.level - 1 - k][idx].x * beta[off_b - k];
        sum1_beta.y += A_rhs[p.level - 1 - k][idx].y * beta[off_b - k];
    }
    //sum1_beta = hipCmul(sum1_beta, make_hipDoubleComplex(p.delta_t, 0.0));
    //A[p.level - tlev][idx] = hipCmul(hipCadd(sum1_alpha, sum1_beta), temp_div);
    A[p.level - tlev][idx].x = (sum1_alpha.x + p.delta_t * sum1_beta.x) * temp_div;
    A[p.level - tlev][idx].y = (sum1_alpha.y + p.delta_t * sum1_beta.y) * temp_div;
}


__global__
void d_integrate_stiff_debug(cuda::cmplx_t** A, cuda::cmplx_t** A_rhs, cuda::real_t* alpha, cuda::real_t* beta, cuda::stiff_params_t p, uint tlev)
{
    //const uint col = 1;
    const uint row = 1;
    //const uint idx = row * p.My + col;
    const uint idx = 2;

    uint off_a = (tlev - 2) * p.level + tlev;
    uint off_b = (tlev - 2) * (p.level - 1) + tlev - 1;
    cuda::real_t kx = cuda::TWOPI * cuda::real_t(row) / p.length_x;
    cuda::real_t ky = cuda::TWOPI * cuda::real_t(row) / p.length_y;
    printf("delta_t = %f, diff = %f\n", p.delta_t, p.diff);
    cuda::cmplx_t sum1_alpha = make_hipDoubleComplex(0.0, 0.0);
    cuda::cmplx_t sum1_beta = make_hipDoubleComplex(0.0, 0.0);
    //cuda::cmplx_t temp_div = make_hipDoubleComplex(1. / (alpha[(tlev - 2) * p.level].x + p.delta_t * p.diff * (kx * kx + ky * ky)), 0.0);
    cuda::real_t temp_div = 1. / (alpha[(tlev - 2) * p.level] + p.delta_t * p.diff * (kx * kx + ky * ky));

    printf("\ttlev = %d, off_a = %d, off_b = %d\n", tlev, off_a, off_b);
    for(uint k = 1; k < tlev; k++)
    {
        printf("\ttlev=%d,k=%d\t %f * A[%d] + dt * %f * A_R[%d]\n", tlev, k, alpha[off_a - k], p.level - k, beta[off_b - k], p.level - 1 - k);
        printf("\ttlev=%d, k = %d\t sum_alpha += %f * (%f, %f)\n", tlev, k, alpha[off_a - k], (A[p.level -k][idx]).x, (A[p.level -k][idx]).y);
        printf("\ttlev=%d, k = %d\t sum_beta+= %f * (%f, %f)\n", tlev, k, beta[off_b - k], (A_rhs[p.level - 1 - k][idx]).x, (A_rhs[p.level - 1 - k][idx]).y);
        //sum1_alpha = hipCadd(sum1_alpha, hipCmul(A[p.level - k][idx], alpha[off_a - k]));
        //sum1_beta = hipCadd(sum1_beta, hipCmul(A_rhs[p.level - 1 - k][idx], beta[off_b - k]));
        sum1_alpha.x += (A[p.level - k][idx]).x * alpha[off_a - k];
        sum1_alpha.y += A[p.level - k][idx].y * alpha[off_a - k];
        sum1_beta.x += A_rhs[p.level - 1 - k][idx].x * beta[off_b - k];
        sum1_beta.y += A_rhs[p.level - 1 - k][idx].y * beta[off_b - k];
    }
    //sum1_beta = hipCmul(sum1_beta, make_hipDoubleComplex(p.delta_t, 0.0));
    //sum1_beta.x = sum1_beta.x * p.delta_t;
    //sum1_beta.y = sum1_beta.y * p.delta_t;
    A[p.level - tlev][idx].x = (sum1_alpha.x + p.delta_t * sum1_beta.x) * temp_div;
    A[p.level - tlev][idx].y = (sum1_alpha.y + p.delta_t * sum1_beta.y) * temp_div;
    //A[p.level - tlev][idx] = hipCmul(hipCadd(sum1_alpha, sum1_beta), temp_div);
    printf("\ttlev=%d, computing A[%d], gamma_0 = %f\n", tlev, p.level - tlev, alpha[(tlev - 2) * p.level]);
    printf("sum1_alpha = (%f, %f)\tsum1_beta = (%f, %f)\t", sum1_alpha.x, sum1_alpha.y, sum1_beta.x, sum1_beta.y);
    printf("temp_div = %f\n", temp_div); 
    printf("A[%d][%d] = (%f, %f)\n", p.level - tlev, idx, A[p.level - tlev][idx].x, A[p.level - tlev][idx].y);
}


/*
 *
 * Kernels to compute non-linear operators
 *
 */


// Poisson brackt: theta_x * strmf_y - theta_x * strmf_y
__global__
void d_pbracket(cuda::real_t* theta_x, cuda::real_t* theta_y, cuda::real_t* strmf_x, cuda::real_t* strmf_y, cuda::real_t* out, uint Nx, uint My)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;

    if ((col >= My) || (row >= Nx))
       return;
    out[idx] = theta_x[idx] * strmf_y[idx] - theta_y[idx] * strmf_x[idx];
}


// RHS for logarithmic density field:
// theta_x * strmf_x - theta_y * strmf_x + diff * (theta_x^2 + theta_y^2)
__global__
void d_theta_rhs_log(cuda::real_t* theta_x, cuda::real_t* theta_y, cuda::real_t* strmf_x, cuda::real_t* strmf_y, cuda::real_t diff, cuda::real_t* tmp_arr, uint Nx, uint My)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
       return;

    tmp_arr[idx] = theta_x[idx] * strmf_y[idx] - theta_y[idx] * strmf_x[idx] + diff * (theta_x[idx] * theta_x[idx] + theta_y[idx] * theta_y[idx]);
}

__global__
void d_theta_rhs_hw(cuda::cmplx_t* theta_rhs_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* theta_hat, cuda::cmplx_t* strmf_y_hat, cuda::real_t C, uint Nx, uint My)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
        return;
    theta_rhs_hat[idx].x += C * (strmf_hat[idx].x  - theta_hat[idx].x) - strmf_y_hat[idx].x;
    theta_rhs_hat[idx].y += C * (strmf_hat[idx].y  - theta_hat[idx].y) - strmf_y_hat[idx].y;
}


__global__
void d_theta_rhs_hw_debug(cuda::cmplx_t* theta_rhs_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* theta_hat, cuda::cmplx_t* strmf_y_hat, cuda::real_t C, uint Nx, uint My)
{
    //const uint col = 2;
    //const uint row = 0;
    //const uint idx = row * My + col;
    const uint idx = 2;

    cuda::cmplx_t dummy = theta_rhs_hat[idx]; 
    dummy.x = theta_rhs_hat[idx].x + C * (strmf_hat[idx].x - theta_hat[idx].x) - strmf_y_hat[idx].x;
    dummy.y = theta_rhs_hat[idx].y + C * (strmf_hat[idx].y - theta_hat[idx].y) - strmf_y_hat[idx].y;
    printf("d_theta_rhs_hw_debug: initially: theta_rhs_hat[%d] = (%f, %f)\t", idx, theta_rhs_hat[idx].x, theta_rhs_hat[idx].y);
    printf("--> theta_rhs_hat[%d] = (%f, %f)\tC = %f, theta_hat = (%f, %f), strmf_hat =(%f, %f), strmf_y_hat=(%f,%f)\n" ,
            idx, dummy.x, dummy.y, C, (theta_hat[idx]).x, (theta_hat[idx]).y, (strmf_hat[idx]).x, (strmf_hat[idx]).y, 
            (strmf_y_hat[idx]).x, (strmf_y_hat[idx]).y); 
}


// RHS for vorticity eq, interchange turbulence
// RHS = RHS - int * theta_y - sdiss * strmf - collfric * omega
__global__
void d_omega_ic_sec1(cuda::cmplx_t* theta_y_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* omega_hat, cuda::real_t ic, cuda::real_t sdiss, cuda::real_t cfric, cuda::cmplx_t* out, uint Nx, uint My)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
       return;

    out[idx].x -= ic * theta_y_hat[idx].x - sdiss * strmf_hat[idx].x - cfric * omega_hat[idx].x;
    out[idx].y -= ic * theta_y_hat[idx].y - sdiss * strmf_hat[idx].y - cfric * omega_hat[idx].y;
}


__global__
void d_omega_rhs_hw(cuda::cmplx_t* omega_rhs_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* theta_hat, cuda::real_t C, uint Nx, uint My)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if ((col >= My) || (row >= Nx))
       return;

    omega_rhs_hat[idx].x += C * (strmf_hat[idx].x - theta_hat[idx].x);
    omega_rhs_hat[idx].y += C * (strmf_hat[idx].y - theta_hat[idx].y);
}


__global__
void d_omega_rhs_hw_debug(cuda::cmplx_t* omega_rhs_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* theta_hat, cuda::real_t C, uint Nx, uint My)
{
    const uint col = 2;
    const uint row = 0;
    const uint idx = row * My + col;

    cuda::cmplx_t dummy;
    dummy.x = omega_rhs_hat[idx].x + C * (strmf_hat[idx].x - theta_hat[idx].x);
    dummy.y = omega_rhs_hat[idx].y + C * (strmf_hat[idx].y - theta_hat[idx].y);
    printf("d_omega_rhs_hw_debug: omega_rhs_hat[%d] = (%f, %f)\tC = %f, strmf_hat = (%f, %f), theta_hat =(%f, %f), strmf_y_hat=(%f,%f)\n" ,
            idx, dummy.x, dummy.y, C, strmf_hat[idx].x, strmf_hat[idx].y, theta_hat[idx].x, theta_hat[idx].y);
}


__global__
void d_omega_ic_dummy(cuda::cmplx_t* theta_y_hat, cuda::cmplx_t* strmf_hat, cuda::cmplx_t* omega_hat, cuda::cmplx_t ic, cuda::cmplx_t sdiss, cuda::cmplx_t cfric, cuda::cmplx_t* out, uint Nx, uint My)
{
    //const uint col = 0;
    //const uint row = 0;
    //const uint idx = 1;
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * My + col;
    if((col >= My) || (row >= Nx))
        return;
    //printf("d_omega_ic_dummy\n");
    //printf("theta_y_hat = (%f, %f), strmf_hat = (%f, %f), omega_hat = (%f, %f)\n", (theta_y_hat[idx]).x, (theta_y_hat[idx]).y, (strmf_hat[idx]).x, (strmf_hat[idx]).y, (omega_hat[idx]).x, (omega_hat[idx]).x);
    //printf("ic = (%f, %f), sdiss = (%f, %f), cfric = (%f, %f)\n", ic.x, ic.y, sdiss.x, sdiss.y, cfric.x, cfric.y);
    //printf("omega_rhs_hat = (%f, %f)\n", (out[idx]).x, (out[idx]).y);
    //cuda::cmplx_t part1 = hipCmul(ic, theta_y_hat[idx]);
    //out[idx] = hipCsub(out[idx], part1);
    //printf("part1 = (%f, %f), out[idx] = (%f, %f)", part1.x, part1.y, out[idx].x, out[idx].y);

    //cuda::cmplx_t part2 = hipCmul(sdiss, strmf_hat[idx]);
    //out[idx] = hipCsub(out[idx], part2);
    //
    //cuda::cmplx_t part3 = hipCmul(cfric, omega_hat[idx]);
    //out[idx] = hipCsub(out[idx], part3);

    //printf("omega_rhs_hat = (%f, %f)\n", (out[idx]).x, (out[idx]).y);
    out[idx] = make_hipDoubleComplex(double(col), double(row));
}

/*****************************************************************************
 *
 * Function implementation
 *
 ****************************************************************************/


void slab_cuda :: d_dx(twodads::field_k_t src_name, twodads::field_k_t dst_name, uint tlev)
{
    cuda_array<cuda::cmplx_t>* arr_in = get_field_by_name(src_name);
    cuda_array<cuda::cmplx_t>* arr_out = get_field_by_name(dst_name);
    
    const uint my21 = My / 2 + 1;
    double Lx = config.get_deltax() * double(Nx);
    //dim3 grid_dx_half(Nx / 2, arr_in -> get_grid().y);
    //dim3 grid_dx_single(1, arr_in -> get_grid().y);

    //d_d_dx_lo<<<grid_dx_half, arr_in -> get_block()>>>(arr_in -> get_array_d(0), arr_out -> get_array_d(0), Nx, My / 2 + 1, Lx);
    //d_d_dx_mid<<<grid_dx_single, arr_in -> get_block()>>>(arr_in -> get_array_d(0), arr_out -> get_array_d(0), Nx, My / 2 + 1, Lx);
    //d_d_dx_up<<<grid_dx_half, arr_in -> get_block()>>>(arr_in -> get_array_d(0), arr_out -> get_array_d(0), Nx, My / 2 + 1, Lx);
    d_d_dx_lo<<<grid_dx_half, block_my21_sec1>>>(arr_in -> get_array_d(tlev), arr_out -> get_array_d(0), Nx, my21, Lx);
    d_d_dx_mid<<<grid_dx_single, block_my21_sec1>>>(arr_in -> get_array_d(tlev), arr_out -> get_array_d(0), Nx, my21, Lx);
    d_d_dx_up<<<grid_dx_half, block_my21_sec1>>>(arr_in -> get_array_d(tlev), arr_out -> get_array_d(0), Nx, my21, Lx);
    hipDeviceSynchronize();    
}


void slab_cuda :: d_dy(twodads::field_k_t src_name, twodads::field_k_t dst_name, uint tlev)
{
    cuda_array<cuda::cmplx_t>* arr_in = get_field_by_name(src_name);
    cuda_array<cuda::cmplx_t>* arr_out = get_field_by_name(dst_name);

    const uint my21 = My / 2 + 1;
    double Ly = config.get_lengthy();
    //dim3 block_single(cuda::cuda_blockdim_nx);
    //dim3 grid_single(Nx / cuda::cuda_blockdim_nx);

    d_d_dy_lo<<<arr_in -> get_grid(), arr_out -> get_block()>>>(arr_in -> get_array_d(tlev), arr_out -> get_array_d(0), Nx, my21, Ly);
    //d_d_dy_up<<<grid_single, block_single>>>(arr_in -> get_array_d(0), arr_out -> get_array_d(0), Nx, My / 2 + 1, Ly);
    d_d_dy_up<<<grid_my21_sec2, block_my21_sec2>>>(arr_in -> get_array_d(tlev), arr_out -> get_array_d(0), Nx, my21, Ly);
    hipDeviceSynchronize();    
}


void slab_cuda :: inv_laplace(twodads::field_k_t src_name, twodads::field_k_t dst_name, uint t_in)
{
    cuda_array<cuda::cmplx_t>* arr_in = get_field_by_name(src_name);
    cuda_array<cuda::cmplx_t>* arr_out = get_field_by_name(dst_name);

    //const uint Nx = config.get_nx();
    //const uint My = config.get_my() / 2 + 1;
    const uint my21 = My / 2 + 1;
    const double inv_Lx2 = 1. / (config.get_lengthx() * config.get_lengthx());
    const double inv_Ly2 = 1. / (config.get_lengthy() * config.get_lengthy());

#ifdef DEBUG
    cout << "slab_chda::inv_laplace(...)\n";
    cout << "block_sec12 = (" << block_sec12.x << ", " << block_sec12.y << ")\t";
    cout << "grid_sec1 = (" << grid_sec1.x << ", " << grid_sec1.y << ")\n";
    cout << "grid_sec2 = (" << grid_sec2.x << ", " << grid_sec2.y << ")\n";
    cout << "grid_sec3 = (" << grid_sec3.x << ", " << grid_sec3.y << ")\n";
    cout << "grid_sec4 = (" << grid_sec4.x << ", " << grid_sec4.y << ")\n";
#endif //DEBUG

    d_inv_laplace_sec1<<<grid_sec1, block_sec12>>>(arr_in -> get_array_d(t_in), arr_out -> get_array_d(0), Nx, my21, inv_Lx2, inv_Ly2);
    d_inv_laplace_sec2<<<grid_sec2, block_sec12>>>(arr_in -> get_array_d(t_in), arr_out -> get_array_d(0), Nx, my21, inv_Lx2, inv_Ly2);
    d_inv_laplace_sec3<<<grid_sec3, block_sec3>>>(arr_in -> get_array_d(t_in), arr_out -> get_array_d(0), Nx, my21, inv_Lx2, inv_Ly2);
    d_inv_laplace_sec4<<<grid_sec4, block_sec4>>>(arr_in -> get_array_d(t_in), arr_out -> get_array_d(0), Nx, my21, inv_Lx2, inv_Ly2);
    d_inv_laplace_zero<<<1, 1>>>(arr_out -> get_array_d(0));
    hipDeviceSynchronize();    
}


void slab_cuda :: integrate_stiff(twodads::dyn_field_t fname, uint tlev)
{
    cuda_array<cuda::cmplx_t>* A = get_field_by_name(fname); 
    cuda_array<cuda::cmplx_t>* A_rhs = get_rhs_by_name(fname); 
    //d_integrate_stiff_debug<<<1, 1>>>(A->get_array_d_t(), A_rhs->get_array_d_t(), d_ss3_alpha, d_ss3_beta, stiff_params, tlev);
    d_integrate_stiff_sec1<<<grid_sec1, block_sec12>>>(A->get_array_d_t(), A_rhs->get_array_d_t(), d_ss3_alpha, d_ss3_beta, stiff_params, tlev);
    d_integrate_stiff_sec2<<<grid_sec2, block_sec12>>>(A->get_array_d_t(), A_rhs->get_array_d_t(), d_ss3_alpha, d_ss3_beta, stiff_params, tlev);
    d_integrate_stiff_sec3<<<grid_sec3, block_sec3>>>(A->get_array_d_t(), A_rhs->get_array_d_t(), d_ss3_alpha, d_ss3_beta, stiff_params, tlev);
    d_integrate_stiff_sec4<<<grid_sec4, block_sec4>>>(A->get_array_d_t(), A_rhs->get_array_d_t(), d_ss3_alpha, d_ss3_beta, stiff_params, tlev);
    hipDeviceSynchronize();
}

void slab_cuda :: theta_rhs_lin(uint t)
{
    d_pbracket<<<grid_nx_my, block_nx_my>>>(theta_x.get_array_d(), theta_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), tmp_array.get_array_d(), Nx, My);
    dft_r2c(twodads::f_tmp, twodads::f_theta_rhs_hat, 0);
    hipDeviceSynchronize();
}


void slab_cuda :: theta_rhs_hw(uint t)
{
    cuda::real_t C = config.get_model_params(1);
    //theta_rhs_hat = make_hipDoubleComplex(0.0, 0.0);
    d_pbracket<<<grid_nx_my, block_nx_my>>>(theta_x.get_array_d(), theta_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), tmp_array.get_array_d(), Nx, My);
    dft_r2c(twodads::f_tmp, twodads::f_theta_rhs_hat, 0);
    //cout << "theta_rhs_hw: theta_rhs_Hat = \n" << theta_rhs_hat << "\n";
    //d_theta_rhs_hw_debug<<<1, 1>>>(theta_rhs_hat.get_array_d(0), strmf_hat.get_array_d(), theta_hat.get_array_d(t), strmf_y_hat.get_array_d(), C, Nx, My / 2 + 1);
    d_theta_rhs_hw<<<grid_my21_sec1, block_my21_sec1>>>(theta_rhs_hat.get_array_d(0), strmf_hat.get_array_d(), theta_hat.get_array_d(t), strmf_y_hat.get_array_d(), C, Nx, My / 2 + 1);
    //hipDeviceSynchronize();
    //cout << "theta_rhs_hw: theta_rhs_Hat = \n" << theta_rhs_hat << "\n";
}


void slab_cuda :: theta_rhs_log(uint t)
{
    d_pbracket<<<grid_nx_my, block_nx_my>>>(theta_x.get_array_d(), theta_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), tmp_array.get_array_d(), Nx, My);
    d_theta_rhs_log<<<grid_nx_my, block_nx_my>>>(theta_x.get_array_d(), theta_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), stiff_params.diff, tmp_array.get_array_d(), Nx, My);
    dft_r2c(twodads::f_tmp, twodads::f_theta_rhs_hat, 0);
    hipDeviceSynchronize();
}


void slab_cuda :: omega_rhs_hw(uint t)
{
    cuda::real_t C = config.get_model_params(1);
    //omega_rhs_hat = make_hipDoubleComplex(0.0, 0.0);
    d_pbracket<<<grid_nx_my, block_nx_my>>>(omega_x.get_array_d(), omega_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), tmp_array.get_array_d(), Nx, My);
    //dft_r2c(twodads::f_tmp, twodads::f_theta_rhs_hat, 0);
    //d_omega_rhs_hw_debug<<<1, 1>>>(omega_rhs_hat.get_array_d(0), strmf_hat.get_array_d(), theta_hat.get_array_d(t), C, Nx, My / 2 + 1);
    d_omega_rhs_hw<<<grid_my21_sec1, block_my21_sec1>>>(omega_rhs_hat.get_array_d(0), strmf_hat.get_array_d(), theta_hat.get_array_d(t), C, Nx, My / 2 + 1);
    //hipDeviceSynchronize();
}


void slab_cuda::omega_rhs_ic(uint t)
{
    // Compute Poisson bracket in real space, use full grid/block
    d_pbracket<<<grid_nx_my, block_nx_my>>>(theta_x.get_array_d(), theta_y.get_array_d(), strmf_x.get_array_d(), strmf_y.get_array_d(), tmp_array.get_array_d(), Nx, My);
    dft_r2c(twodads::f_tmp, twodads::f_tmp_hat, 0);
    // Convert model parameters to complex numbers
    cuda::real_t ic = config.get_model_params(2); 
    cuda::real_t sdiss = config.get_model_params(3);
    cuda::real_t cfric = config.get_model_params(4);
#ifdef DEBUG
    cout << "omega_rhs\n";
    cout << "ic = (" << ic.x << ", " << ic.y << "), sdiss = (" << sdiss.x << ", " << sdiss.y << "), cfric = (" << cfric.x << ", " << cfric.y << ")\n";
    cout << "grid = (" << theta_hat.get_grid().x << ", " << theta_hat.get_grid().y << "), block = (" << theta_hat.get_block().x << ", " << theta_hat.get_block().y << ")\n";
#endif //DEBUG
    //d_omega_ic_dummy<<<grid_my21_sec1, block_my21_sec1>>>(theta_y_hat.get_array_d(), strmf_hat.get_array_d(), omega_hat.get_array_d(0), ic, sdiss, cfric, omega_rhs_hat.get_array_d(0), Nx, My / 2 + 1);
    d_omega_ic_sec1<<<grid_my21_sec1, block_my21_sec1>>>(theta_y_hat.get_array_d(0), strmf_hat.get_array_d(0), omega_hat.get_array_d(0), ic, sdiss, cfric, omega_rhs_hat.get_array_d(0), Nx, My / 2 + 1);
    hipDeviceSynchronize();
}



void slab_cuda::dump_address()
{
    cout << "\nCompiled with NVCC\n";
    cout << "slab_cuda::dump_address()\n";
    cout << "\tconfig at " << (void*) &config << "\n";
    cout << "\tNx at " << (void*) &Nx << "\n";
    cout << "\tMy at " << (void*) &My << "\n";
    cout << "\ttlevs at " << (void*) &tlevs << "\n";
    cout << "\tplan_r2c at " << (void*) &plan_r2c << "\n";
    cout << "\tplan_c2r at " << (void*) &plan_c2r << "\n";
    cout << "\tslab_output at " << (void*) &slab_output << "\n";
    cout << "\ttheta at " << (void*) &theta << "\n";
    cout << "\ttheta_x at " << (void*) &theta_x << "\n";
    cout << "\ttheta_y at " << (void*) &theta_y << "\n";
    cout << "\tslab_output at " << (void*) &slab_output << "\n";
    cout << "\tstiff_params at " << (void*) &stiff_params << "\n";

}


// End of file slab_cuda2.cu
