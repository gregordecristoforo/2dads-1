#include "hip/hip_runtime.h"
/*
 * CUDA specific initialization function
 *
 */

#include <iostream>
#include <vector>
#include "include/initialize.h"


__global__ 
void d_init_sine(cuda::real_t* array, cuda::slab_layout_t layout, double kx, double ky)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row  = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.Nx) || (row >= layout.My))
        return;
    array[idx] = sin(kx * x) + sin(ky * y);
}


__global__
void d_init_exp(cuda::real_t* array, cuda::slab_layout_t layout, cuda::init_params_t params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params.i1 + params.i2 * exp( -(x - params.i3) * (x - params.i3) / (2.0 * params.i4 * params.i4) 
                                              -(y - params.i5) * (y - params.i5) / (2.0 * params.i6 * params.i6));
}


__global__
void d_init_exp_log(cuda::real_t* array, cuda::slab_layout_t layout, cuda::init_params_t params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params.i1 + params.i2 * exp( -(x - params.i3) * (x - params.i3) / (2.0 * params.i4 * params.i4) 
                                              -(y - params.i5) * (y - params.i5) / (2.0 * params.i6 * params.i6));
    array[idx] = log(array[idx]);
}


__global__
void d_init_lapl(cuda::real_t* array, cuda::slab_layout_t layout, cuda::init_params_t params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = exp(- 0.5 * (x * x + y * y)/(params.i4 * params.i4)) / (params.i4 * params.i4) * 
                 ((x * x + y * y)/(params.i4 * params.i4) - 2.0);
}


/// Initialize gaussian profile around a single mode
__global__
void d_init_mode_exp(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, double modex, double modey, double sigma)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * (layout.My / 2 + 1) + col;
    if ((col >= layout.My / 2 + 1) || (row >= layout.Nx))
        return;
    double n = double(row);
    double m = double(col);
    double damp = exp ( -((n-modex)*(n-modex) / sigma) - ((m-modey)*(m-modey) / sigma) ); 
    double phase = 0.56051 * 2.0 * cuda::PI;  
    
    cuda::cmplx_t foo(damp * amp * cos(phase), damp * amp * sin(phase));
    array[idx] = foo;
    //printf("sigma = %f, re = %f, im = %f\n", sigma, array[idx].x, array[idx].y);
}


/// Initialize a single mode pointwise
__global__
void d_init_mode(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, uint row, uint col)
{
    const uint idx = row * (layout.My / 2 + 1) + col;
    const double phase = 0.56051 * cuda::TWOPI;
    CuCmplx<cuda::real_t> foo(amp * cos(phase), amp * sin(phase));
    array[idx] = foo;
    printf("d_init_mode: mode(%d, %d) at idx = %d = (%f, %f)\n",
            row, col, idx, cos(phase), sin(phase));
}


/// Initialize all modes to given value
__global__
void d_init_all_modes(cuda::cmplx_t* array, cuda::slab_layout_t layout, double real, double imag)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * (layout.My / 2 + 1) + col;
    if ((col >= layout.My / 2 + 1) || (row >= layout.Nx))
        return;

    array[idx] = cuda::cmplx_t(real, imag);
}


/// Initialize sinusoidal profile
//void init_simple_sine(cuda_array<cuda::real_t, cuda::real_t>* arr, 
void init_simple_sine(cuda_arr_real* arr,
        vector<double> initc,
        cuda::slab_layout_t layout)
{
    dim3 grid = arr -> get_grid();
    dim3 block = arr -> get_block();

    const double kx = initc[0] * cuda::TWOPI / double(layout.delta_x * double(arr -> get_nx()));
    const double ky = initc[1] * cuda::TWOPI / double(layout.delta_y * double(arr -> get_my()));

    d_init_sine<<<grid, block>>>(arr -> get_array_d(0), layout, kx, ky);
    hipDeviceSynchronize();
}


/// Initialize field with a guassian profile
void init_gaussian(cuda_array<cuda::real_t, cuda::real_t>* arr,
        vector<double> initc,
        cuda::slab_layout_t layout,
        bool log_theta)
{
    cuda::init_params_t init_params;

    init_params.i1 = (initc.size() > 0) ? initc[0] : 0.0;
    init_params.i2 = (initc.size() > 1) ? initc[1] : 0.0;
    init_params.i3 = (initc.size() > 2) ? initc[2] : 0.0;
    init_params.i4 = (initc.size() > 3) ? initc[3] : 0.0;
    init_params.i5 = (initc.size() > 4) ? initc[4] : 0.0;
    init_params.i6 = (initc.size() > 5) ? initc[5] : 0.0;

    if (log_theta)
        d_init_exp_log<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, init_params);
    else
        d_init_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, init_params);
    hipDeviceSynchronize();
}


/// Initialize real field with nabla^2 exp(-(x-x0)^2/ (2. * sigma^2) - (y - y0)^2 / (2. * sigma_y^2)
void init_invlapl(cuda_array<cuda::real_t, cuda::real_t>* arr,
        vector<double> initc,
        cuda::slab_layout_t layout)
{

    cuda::init_params_t init_params;
    init_params.i1 = (initc.size() > 1) ? initc[0] : 0.0;
    init_params.i2 = (initc.size() > 2) ? initc[1] : 0.0;
    init_params.i3 = (initc.size() > 3) ? initc[2] : 0.0;
    init_params.i4 = (initc.size() > 4) ? initc[3] : 0.0;
    init_params.i5 = (initc.size() > 5) ? initc[4] : 0.0;
    init_params.i6 = (initc.size() > 6) ? initc[5] : 0.0;

    d_init_lapl<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, init_params);
    hipDeviceSynchronize();
}

// Initialize all modes with constant value
void init_all_modes(cuda_arr_cmplx* arr, vector<double> initc, cuda::slab_layout_t layout, uint tlev)
{
    double real = initc[0];
    double imag = initc[1];
d_init_all_modes<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(tlev), layout, real, imag);
}


/// Initialize single mode from input.ini
/// Use 
/// init_function  = *_mode
/// initial_conditions = amp_0 kx_0 ky_0 sigma_0   amp_1 kx_1 ky_1 sigma_1 .... amp_N kx_N ky_N sigma_N 
/// kx -> row, ky -> column. 
/// kx is the x-mode number: kx = 0, 1, ...Nx -1
/// ky is the y-mode number: ky = 0... My / 2
void init_mode(cuda_array<cuda::cmplx_t, cuda::real_t>* arr,
        vector<double> initc,
        cuda::slab_layout_t layout,
        uint tlev)
{
    const unsigned int num_modes = initc.size() / 4;
    (*arr) = CuCmplx<cuda::real_t>(0.0, 0.0);

#ifdef DEBUG
    cout << "Initializing modes:\n";
#endif
    for(uint n = 0; n < num_modes; n++)
    {
#ifdef DEBUG
        cout << "mode " << n << ": amp=" << initc[4*n] << " kx=" << initc[4*n+1] << ", ky=" << initc[4*n+2] << ", sigma=" << initc[4*n+3] << "\n";
#endif
        d_init_mode_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(tlev), layout, initc[4*n], initc[4*n+1], initc[4*n+2], initc[4*n+3]);
        //d_init_mode<<<1, 1>>>(arr -> get_array_d(0), layout, initc[4*n], uint(initc[4*n+1]), uint(initc[4*n+2])); 
    }
}

// End of file initialize.cu
