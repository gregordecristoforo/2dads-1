#include "hip/hip_runtime.h"
/*
 * CUDA specific initialization function
 *
 */

#include <iostream>
#include <vector>
#include "include/cuda_types.h"
#include "include/cuda_array3.h"


__global__ 
void d_init_sine(cuda::real_t* array, cuda::slab_layout_t layout, double kx, double ky)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row  = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.Nx) || (row >= layout.My))
        return;
    array[idx] = sin(kx * x) + sin(ky * y);
}


__global__
void d_init_exp(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params[0] + params[1] * exp( -(x - params[2]) * (x - params[2]) / (2.0 * params[3] * params[3]) 
                                              -(y - params[4]) * (y - params[4]) / (2.0 * params[5] * params[5]));
}


__global__
void d_init_exp_log(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params[0] + params[1] * exp( -(x - params[2]) * (x - params[2]) / (2.0 * params[3] * params[3]) 
                                              -(y - params[4]) * (y - params[4]) / (2.0 * params[5] * params[5]));
    array[idx] = log(array[idx]);
}


__global__
void d_init_lapl(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = exp(- 0.5 * (x * x + y * y)/(params[3] * params[3])) / (params[3] * params[3]) * 
                 ((x * x + y * y)/(params[3] * params[3]) - 2.0);
}


/// Initialize gaussian profile around a single mode
__global__
//void d_init_mode_exp(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, double modex, double modey, double sigma)
void d_init_mode_exp(CuCmplx<cuda::real_t>* array, cuda::slab_layout_t layout, double amp, double modex, double modey, double sigma)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    if ((col >= layout.My) || (row >= layout.Nx))
        return;
    double n = double(row);
    double m = double(col);
    double damp = exp ( -((n-modex)*(n-modex) / sigma) - ((m-modey)*(m-modey) / sigma) ); 
    double phase = 0.56051 * 2.0 * cuda::PI;  
    
    //array[idx] = make_hipDoubleComplex(damp * amp * cos(phase), damp * amp * sin(phase));
    CuCmplx<cuda::real_t> foo(damp * amp * cos(phase), damp * amp * sin(phase));
    array[idx] = foo;
    //printf("sigma = %f, re = %f, im = %f\n", sigma, array[idx].x, array[idx].y);
}


/// Initialize a single mode pointwise
__global__
void d_init_mode(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, uint col, uint row)
{
    const uint idx = row * layout.My + col;
    const double phase = 0.56051 * cuda::TWOPI;
    CuCmplx<cuda::real_t> foo(amp * cos(phase), amp * sin(phase));
    array[idx] = foo;
    printf("d_init_mode: mode(%d, %d) at idx = %d = (%f, %f)\n",
            row, col, idx, cos(phase), sin(phase));
}


/// Initialize sinusoidal profile
void init_simple_sine(cuda_array<cuda::real_t, cuda::real_t>* arr, 
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    cout << "init_simple_sine()\n";
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    dim3 grid = arr -> get_grid();
    dim3 block = arr -> get_block();

    const double kx = initc[0] * cuda::TWOPI / double(layout.delta_x * double(arr -> get_nx()));
    const double ky = initc[1] * cuda::TWOPI / double(layout.delta_y * double(arr -> get_my()));

    //d_init_sine<<<grid, block>>>(arr -> get_array_d(0), layout, d_params);
    d_init_sine<<<grid, block>>>(arr -> get_array_d(0), layout, kx, ky);
    hipDeviceSynchronize();
}


/// Initialize field with a guassian profile
void init_gaussian(cuda_array<cuda::real_t, cuda::real_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo,
        bool log_theta)
{
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    double* params = initc.data();
    double* d_params;

    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    if (log_theta)
    {
        cout << "Initializing logarithmic theta\n";
        d_init_exp_log<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    }
    else
    {
        cout << "Initializing theta\n";
        d_init_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    }
    cout << "initc = (" << params[0] << ", " << params[1] << ", " << params[2] << ", ";
    cout << params[3] << ", " << params[4] << ", " << params[5] << ")\n";
    hipDeviceSynchronize();
}



/// Initialize real field with nabla^2 exp(-(x-x0)^2/ (2. * sigma^2) - (y - y0)^2 / (2. * sigma_y^2)
void init_invlapl(cuda_array<cuda::real_t, cuda::real_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    cout << "init_invlapl\n";
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    double* params = initc.data();
    double* d_params;
    cout << "initc = (" << params[0] << ", " << params[1] << ", " << params[2] << ", ";
    cout << params[3] << ", " << params[4] << ", " << params[5] << ")\n";

    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    d_init_lapl<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    hipDeviceSynchronize();
}



void init_mode(cuda_array<CuCmplx<cuda::real_t>, cuda::real_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    // We call arr -> get_my() which is alreade reduced to My/2+1 from the slab since arr is of
    // type cuda_array<cuda::cmplx_t>
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    const unsigned int num_modes = initc.size() / 4;
    
    //(*arr) = make_hipDoubleComplex(0.0, 0.0);
    (*arr) = CuCmplx<cuda::real_t>(0.0, 0.0);
    for(uint n = 0; n < num_modes; n++)
    {
        cout << "mode " << n << ": amp=" << initc[4*n] << " ky=" << initc[4*n+1] << ", kx=" << initc[4*n+2] << ", sigma=" << initc[4*n+3] << "\n";
        d_init_mode_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, initc[4*n], initc[4*n+1], initc[4*n+2], initc[4*n+3]);
    }
    hipDeviceSynchronize();
}



// End of file initialize.cu
