#include "hip/hip_runtime.h"
/*
 *
 * CUDA specific initialization function
 *
 */

#include <iostream>
#include <vector>
#include "include/cuda_types.h"
#include "include/cuda_array2.h"


__global__ 
void d_init_sine(cuda::real_t* array, cuda::slab_layout_t layout, double* params) 
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row  = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.Nx) || (row >= layout.My))
        return;
    array[idx] = sin(params[0] * cuda::PI * x) + sin(params[1] * cuda::PI * y);
}


__global__
void d_init_exp(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params[0] + params[1] * exp( -(x - params[2]) * (x - params[2]) / (2.0 * params[3] * params[3]) 
                                              -(y - params[4]) * (y - params[4]) / (2.0 * params[5] * params[5]));
}


__global__
void d_init_exp_log(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = params[0] + params[1] * exp( -(x - params[2]) * (x - params[2]) / (2.0 * params[3] * params[3]) 
                                              -(y - params[4]) * (y - params[4]) / (2.0 * params[5] * params[5]));
    array[idx] = log(array[idx]);
}


__global__
void d_init_lapl(cuda::real_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    const double x = layout.x_left + double(row) * layout.delta_x;
    const double y = layout.y_lo + double(col) * layout.delta_y;

    if ((col >= layout.My) || (row >= layout.Nx))
        return;

    array[idx] = exp(- 0.5 * (x * x + y * y)/(params[3] * params[3])) / (params[3] * params[3]) * 
                 ((x * x + y * y)/(params[3] * params[3]) - 2.0);
}


__global__
void d_init_mode_exp(cuda::cmplx_t* array, cuda::slab_layout_t layout, double* params)
{
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idx = row * layout.My + col;
    if ((col >= layout.My) || (row >= layout.Nx))
        return;
    double n = double(row);
    double m = double(col);
    double amplitude = params[0];
    double modex = params[1];
    double modey = params[2];
    double damp = exp ( -((n-modex)*(n-modex) / 0.1) - ((m-modey)*(m-modey) / 0.1) ); 
    double phase = 0.56051 * 2.0 * cuda::PI;  

    array[idx] = make_hipDoubleComplex(damp * amplitude * cos(phase), damp * amplitude * sin(phase));
}


//__global__
//void d_init_mode(cuda::cmplx_t* array, cuda::slab_layout_t layout, double* params)
//{
//    const uint idx = uint(params[2]) * layout.My + uint(params[1]);
//    double phase = 0.56051 * 2.0 * cuda::PI;
//    array[idx] = make_hipDoubleComplex(params[0] * sin(phase), params[0] * cos(phase));
//    printf("d_init_mide: mode (%d, %d) at idx= %d = (%f, %f)\n",
//            uint(params[2]), uint(params[1]), idx, params[0] * sin(phase), params[0] * cos(phase));
//}


__global__
void d_init_mode(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, uint col, uint row)
{
    const uint idx = row * layout.My + col;
    const double phase = 0.56051 * cuda::TWOPI;
    array[idx] = make_hipDoubleComplex(amp * cos(phase), amp * sin(phase));
    printf("d_init_mode: mode(%d, %d) at idx = %d = (%f, %f)\n",
            row, col, idx, cos(phase), sin(phase));
}

void init_simple_sine(cuda_array<cuda::real_t>* arr, 
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    cout << "init_simple_sine()\n";
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    dim3 grid = arr -> get_grid();
    dim3 block = arr -> get_block();

    double* params = initc.data();
    // Copy the parameters for the function to the device
    double* d_params;
    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    d_init_sine<<<grid, block>>>(arr -> get_array_d(0), layout, d_params);
    //d_init_sine<<<1, 1>>>(arr -> get_array_d(), layout, d_params);
    hipDeviceSynchronize();
}


void init_gaussian(cuda_array<cuda::real_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo,
        bool log_theta)
{
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    double* params = initc.data();
    double* d_params;

    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    if (log_theta)
    {
        cout << "Initializing logarithmic theta\n";
        d_init_exp_log<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    }
    else
    {
        cout << "Initializing theta\n";
        d_init_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    }
    cout << "initc = (" << params[0] << ", " << params[1] << ", " << params[2] << ", ";
    cout << params[3] << ", " << params[4] << ", " << params[5] << ")\n";
    hipDeviceSynchronize();
}


void init_invlapl(cuda_array<cuda::real_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    cout << "init_invlapl\n";
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    double* params = initc.data();
    double* d_params;
    cout << "initc = (" << params[0] << ", " << params[1] << ", " << params[2] << ", ";
    cout << params[3] << ", " << params[4] << ", " << params[5] << ")\n";

    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    d_init_lapl<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    hipDeviceSynchronize();
}


void init_mode(cuda_array<cuda::cmplx_t>* arr,
        vector<double> initc,
        const double delta_x,
        const double delta_y,
        const double x_left,
        const double y_lo)
{
    // We call arr -> get_my() which is alreade reduced to My/2+1 from the slab since arr is of
    // type cuda_array<cuda::cmplx_t>
    cuda::slab_layout_t layout = {x_left, delta_x, y_lo, delta_y, arr -> get_nx(), arr -> get_my()};

    //double* params = initc.data();
    const unsigned int num_modes = initc.size() / 3;
    //for(uint i = 1; i < (initc.size()-1); i++)
    //    cout << "params[" << i << "] = " << params[i] << "\n";
    (*arr) = make_hipDoubleComplex(0.0, 0.0);
    for(uint n = 0; n < num_modes; n++)
    {
        cout << "mode " << n << ": amp=" << initc[3*n] << " ky=" << initc[3*n+1] << ", kx=" << initc[3*n+2] << "\n";
        d_init_mode<<<1, 1>>>(arr -> get_array_d(0), layout, initc[3*n], uint(initc[3*n+1]), uint(initc[3*n+2]));
    }

//    gpuErrchk(hipMalloc( (double**) &d_params, initc.size() * sizeof(double)));
//    gpuErrchk(hipMemcpy(d_params, params, sizeof(double) * initc.size(), hipMemcpyHostToDevice));

    //d_init_mode_exp<<<arr -> get_grid(), arr -> get_block()>>>(arr -> get_array_d(0), layout, d_params);
    //(*arr) = make_hipDoubleComplex(0.0, 0.0);
//void d_init_mode(cuda::cmplx_t* array, cuda::slab_layout_t layout, double amp, uint col, uint row)
//    d_init_mode<<<1, 1>>>(arr -> get_array_d(0), layout, d_params);
    //hipDeviceSynchronize();
}



// End of file initialize.cu
