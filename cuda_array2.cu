#include "hip/hip_runtime.h"
/*
 *
 *  Created on: Oct 22, 2013
 *      Author: Ralph Kube
 *
 *  cuda_array2.cu
 *
 *  Implementation of cuda_array
 *
 */

#include <iostream>
#include "include/cuda_array2.h"

using namespace std;

__device__ int d_get_col(void)
{
    return (blockIdx.y * blockDim.y + threadIdx.y);
}


__device__ int d_get_row(void)
{
    return (blockIdx.x * blockDim.x + threadIdx.x);
}

// Template kernel for d_enumerate_d and d_enumerate_c using the ca_val class
template <typename T>
__global__ void d_enumerate(T* array, int Nx, int My)
{
    //const int col = blockIdx.y * blockDim.y + threadIdx.y;
    //const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = d_get_col();
    const int row = d_get_row();
    const int index = row * (My) + col;

    if (index < Nx * My)
    {
        ca_val<T> val;
        val.set(double(index));
        array[index] = val.get();
    }
}


// Template version of d_enumerate_t_x
template <typename T>
__global__ void d_enumerate_t(T** array_t, int t, int Nx, int My)
{
    //const int col = blockIdx.y * blockDim.y + threadIdx.y;
    //const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = d_get_col();
    const int row = d_get_row();
    const int index = row * My + col;

	if (blockIdx.x + threadIdx.x + threadIdx.y == 0)
		printf("blockIdx.x = %d: enumerating at t = %d, at %p(device)\n", blockIdx.x, t, array_t[t]);
    //printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d,  blockIdy.y = %d, blockDim.y = %d, threadIdy.y = %d ,index = %d\n", blockIdx.x, blockDim.x, threadIdx.x,   blockIdx.y, blockDim.y, threadIdx.y, index);
	if (index < Nx * My)
    {
        ca_val<T> val;
        val.set(double(index));
		array_t[t][index] = val.get();
    }
}


template <typename T>
__global__ void d_set_constant_t(T** array_t, T val, int t, int Nx, int My)
{
	//const int col = blockIdx.y * blockDim.y + threadIdx.y;
	//const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = d_get_col();
    const int row = d_get_row();
	const int index = row * My + col;
	if (index < Nx * My)
		array_t[t][index] = val;
}


template <typename T>
__global__ void d_zero_tlev(T** array_t, int t, int Nx, int My)
{
	//const int col = blockIdx.y * blockDim.y + threadIdx.y;
	//const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = d_get_col();
    const int row = d_get_row();
	const int index = row * My + col;
	if (index < Nx * My)
    {
        ca_val<T> val;
        val.set(double(0.0));
		array_t[t][index] = val.get();
    }
}


template <typename T>
__global__ void d_alloc_array_d_t(T** array_d_t, T* array, int tlevs, int Nx, int My)
{
	if (threadIdx.x < tlevs)
	{
		array_d_t[threadIdx.x] = &array[threadIdx.x * Nx * My];
		printf("Device: array_d_t[%d] at %p\n", threadIdx.x, array_d_t[threadIdx.x]);
	}
}


template <typename T>
__global__ void d_advance(T** array_t, int tlevs)
{
	T* tmp = array_t[tlevs - 1];
	unsigned int t = 0;

    /*
	printf("Before:\n");
	for(t = tlevs - 1; t > 0; t--)
		printf("array_t[%d] at %p\n", t, array_t[t]);
	printf("array_t[0] at %p\n", array_t[0]);
    */
	for(t = tlevs - 1; t > 0; t--)
		array_t[t] = array_t[t - 1];
	array_t[0] = tmp;

    /*
	printf("After:\n");
	for(t = tlevs - 1; t > 0; t--)
		printf("array_t[%d] at %p\n", t, array_t[t]);
	printf("array_t[0] at %p\n", array_t[0]);
    */
}


template <typename T>
__global__ void test_alloc(T** array_t, int tlevs)
{
    for(int t = 0; t < tlevs; t++)
        printf("array_x_t[%d] at %p\n", t, array_t[t]);
}


//template <typename T>
__global__ void d_add(cuda::real_t* lhs, cuda::real_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = lhs[idx] + rhs[idx];
}


__global__ void d_add(cuda::cmplx_t* lhs, cuda::cmplx_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = hipCadd(lhs[idx], rhs[idx]);
}


//template <typename T>
__global__ void d_sub(cuda::real_t* lhs, cuda::real_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = lhs[idx] - rhs[idx];
}


__global__ void d_sub(cuda::cmplx_t* lhs, cuda::cmplx_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = hipCsub(lhs[idx], rhs[idx]);
}


//template <typename T>
__global__ void d_mul(cuda::real_t* lhs, cuda::real_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = lhs[idx] * rhs[idx];
}


__global__ void d_mul(cuda::cmplx_t* lhs, cuda::cmplx_t* rhs, int Nx, int My)
{
    const int col = d_get_col();
    const int row = d_get_row();
    const int idx = row * My + col;
    if ((col < My) && (row < Nx))
        lhs[idx] = hipCmul(lhs[idx], rhs[idx]);
}



// Default constructor
template <class T>
cuda_array<T> :: cuda_array(unsigned int t, unsigned int nx, unsigned int my) :
    tlevs(t), Nx(nx), My(my), bounds(tlevs, Nx, My),
    array_d(NULL), array_d_t(NULL),
    array_h(NULL), array_h_t(NULL)
{
    // Determine grid size for kernel launch
    block = dim3(cuda::cuda_blockdim_nx, cuda::cuda_blockdim_my);
    // Testing, use 64 threads in y direction. Thus blockDim.y = 1
    // blockDim.x is the x, one block for each row
    grid = dim3(Nx, 1);
    grid_full = dim3(tlevs * Nx, 1);
    cout << "blockDim=(" << block.x << ", " << block.y << ", " << block.z << ")\n";
    cout << "gridDim=(" << grid.x << ", " << grid.y << ", " << grid.z << ")\n";

    cout << "Array size: Nx=" << Nx << ", My=" << My << ", tlevs=" << tlevs << "\n";
    // Allocate device memory
    size_t nelem = tlevs * Nx * My;
    gpuErrchk(hipMalloc( (void**) &array_d, nelem * sizeof(T)));
    array_h = (T*) calloc(nelem, sizeof(T));

    cout << "Device data at " << array_d << "\t";
    cout << "Host data at " << array_h << "\t";
    cout << nelem << " bytes of data\n";

    // array_[hd]_t is an array of pointers allocated on the host/device respectively
    gpuErrchk(hipMalloc( (void***) &array_d_t, tlevs * sizeof(T*)));
    array_h_t = (T**) calloc(tlevs, sizeof(T*));
    array_d_t_host = (T**) calloc(tlevs, sizeof(T*));
    // array_t[i] points to the i-th time level
    // Set pointers on device
    d_alloc_array_d_t<<<1, tlevs>>>(array_d_t, array_d, tlevs, Nx, My);
    // Update host copy
    gpuErrchk(hipMemcpy(array_d_t_host, array_d_t, sizeof(T*) * tlevs, hipMemcpyDeviceToHost));

    for(unsigned int tl = 0; tl < tlevs; tl++)
    {
        array_h_t[tl] = &array_h[tl * Nx * My];
        cout << "time level " << tl << " at ";
        cout << array_h_t[tl] << "(host)\t";
        cout << array_d_t_host[tl] << "(device)\n";
    }

    cout << "Testing allocation of array_d_t:\n";
    test_alloc<<<1, 1>>>(array_d_t, tlevs);
}

template <class T>
cuda_array<T> :: ~cuda_array(){
    hipFree(array_d);
    free(array_h);
    hipFree(array_d_t);
    free(array_h_t);
    free(array_d_t_host);
}


// Access functions for private members

template <typename T>
void cuda_array<T> :: enumerate_array(const int t)
{
	if (!bounds(t, Nx-1, My-1))
		throw out_of_bounds_err(string("T& cuda_array<T> :: enumerate_array(const int): out of bounds\n"));
	d_enumerate<<<grid, block>>>(array_d, Nx, My);
	hipDeviceSynchronize();
}


template <typename T>
void cuda_array<T> :: enumerate_array_t(const int t)
{
	if (!bounds(t, Nx-1, My-1))
		throw out_of_bounds_err(string("T& cuda_array<T> :: enumerate_array_t(const int): out of bounds\n"));
	d_enumerate_t<<<grid, block >>>(array_d_t, t, Nx, My);
	hipDeviceSynchronize();
}

// Operators
template <typename T>
cuda_array<T>& cuda_array<T> :: operator= (const cuda_array<T>& rhs)
{
    // check bounds
    if (!bounds(rhs.get_tlevs(), rhs.get_nx(), rhs.get_my()))
        throw out_of_bounds_err(string("cuda_array<T>& cuda_array<T> :: operator= (const cuda_array<T>& rhs): out of bounds!"));
    // Check if we assign to ourself
    if ((void*) this == (void*) &rhs)
        return *this;
    
    // Copy data from other array
    gpuErrchk(hipMemcpy(array_d, rhs.get_array_d(), sizeof(T*) * tlevs * Nx * My, hipMemcpyDeviceToDevice));
    // Leave the pointer to the time leves untouched!!!
    return *this;
}

// Set whole array to rhs
template <typename T>
cuda_array<T>& cuda_array<T> :: operator= (const T& rhs)
{
    for(unsigned int t = 0; t < tlevs; t++)
    {
        d_set_constant_t<<<grid, block>>>(array_d_t, rhs, t, Nx, My);
        hipDeviceSynchronize();
    }
    return *this;
}


template <typename T>
cuda_array<T>& cuda_array<T> :: operator+=(const cuda_array<T>& rhs)
{
    if(!bounds(rhs.get_nx(), rhs.get_my()))
        throw out_of_bounds_err(string("cuda_array<T>& cuda_array<T> :: operator+= (const cuda_array<T>& rhs): out of bounds!"));
    if ((void*) this == (void*) &rhs)
        throw operator_err(string("cuda_array<T>& cuda_array<T> :: operator+= (const cuda_array<T>&): RHS and LHS cannot be the same\n"));

    d_add<<<grid, block>>>(array_d, rhs.get_array_d(), Nx, My);
    hipDeviceSynchronize();
    return *this;
}


template <class T>
cuda_array<T>& cuda_array<T> :: set_all(const T& rhs)
{
    cout << "Deprecated, use operator=\n";
	for(unsigned int t = 0; t < tlevs; t++)
	{
		d_set_constant_t<<<grid, block>>>(array_d_t, rhs, t, Nx, My);
		hipDeviceSynchronize();
	}
	return *this;
}


template <class T>
T& cuda_array<T> :: operator()(unsigned int t, unsigned int n, unsigned int m)
{
	if (!bounds(t, n, m))
		throw out_of_bounds_err(string("T& cuda_array<T> :: operator()(uint, uint, uint): out of bounds\n"));
	return (*(array_h_t[t] + address(n, m)));
}


template <class T>
T cuda_array<T> :: operator()(unsigned int t, unsigned int n, unsigned int m) const
{
	if (!bounds(t, n, m))
		throw out_of_bounds_err(string("T cuda_array<T> :: operator()(uint, uint, uint): out of bounds\n"));
	return (*(array_h_t[t] + address(n, m)));
}


template <class T>
void cuda_array<T> :: advance()
{
	//Advance array_d_t pointer on device
	d_advance<<<1, 1>>>(array_d_t, tlevs);
	hipDeviceSynchronize();
    // Zero out last time level for visualization purpose
    d_zero_tlev<<<grid, block>>>(array_d_t, tlevs - 1, Nx, My); 
    // Update array_d_t_host
    gpuErrchk(hipMemcpy(array_d_t_host, array_d_t, sizeof(T*) * tlevs, hipMemcpyDeviceToHost));
}



template <class T>
string cuda_array<T> :: cout_wrapper(const T& val) const
{
	stringstream ss;
	ss << val;
	return ss.str();
}


template <>
string cuda_array<cuda::cmplx_t> :: cout_wrapper(const cuda::cmplx_t& val) const
{
	stringstream ss;
	ss << "(" << hipCreal(val) << ", " << hipCimag(val) << "i)";
	return ss.str();
}


// The array is contiguous, so use only one memcpy
template <class T>
void cuda_array<T> :: copy_device_to_host() {
    const size_t size_line = Nx * My * tlevs * sizeof(T);
    cout << "Copying " << size_line << " bytes to " << array_h << " (host) from ";
    cout << array_d << " (device)\n";
    gpuErrchk(hipMemcpy(array_h, array_d, size_line, hipMemcpyDeviceToHost));
}


template class cuda_array<cuda::cmplx_t>;
template class cuda_array<cuda::real_t>;

// End of file cuda_array.cpp
